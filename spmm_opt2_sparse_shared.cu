#include "hip/hip_runtime.h"
#include "spmm_opt2_sparse_shared.h"
#include "data.h"
#include <string>
#include <iostream>
#define CONSTINT const int

using namespace std;

extern string base_dir, graph;

const int WARPS_PER_BLOCK = 12;

#define DIM_MUL(x) ((x + 31) / 32) * 32

__global__ void spmm_kernel_opt2_sparse_shared(const int *_warp4, const int *idx, const float *val, const float *vin_data, const int *vin_selector, float *vout, const int num_v, const int num_e, const int feat_in, const int dim_sparse, const int num_warps)
{
    const int4 *warp4 = reinterpret_cast<const int4 *>(_warp4);
    extern __shared__ float out_cache[];

    const int total_tid = blockIdx.x * blockDim.x + threadIdx.x; 
    const int total_warpid = total_tid / dim_sparse; 
    const int laneid = threadIdx.x % dim_sparse;  
    const int wid = threadIdx.x / dim_sparse;
    
    const int4 w_info = warp4[total_warpid];
    CONSTINT warp_row = w_info.x;
    CONSTINT warp_loc = w_info.y;
    CONSTINT warp_len = w_info.z;

#pragma unroll
    // for (int ext = 0; ext < (feat_in + 31) / 32; ext++)
    // {
    //     out_cache[wid * DIM_MUL(feat_in) + laneid + ext * 32] = 0;
    // }
    for (int ext = 0; ext < (feat_in + dim_sparse - 1) / dim_sparse; ext++)
    {
        out_cache[threadIdx.x + ext * blockDim.x] = 0;
    }
    if (total_warpid >= num_warps )
        return; 
    
    __syncthreads();
    

    float tmp = 0;
#pragma unroll
    for (int i = 0; i < warp_len; i++)
    {
        int nz_loc = warp_loc + i;
        float left_val = __ldg(val + nz_loc);
        int right_loc = __ldg(idx + nz_loc) * dim_sparse + laneid;
        float right_val = vin_data[right_loc];
        // atomicAdd(&vout[warp_row * feat_in + __ldg(vin_selector + right_loc)], left_val * right_val);
        out_cache[wid * DIM_MUL(feat_in) + __ldg(vin_selector + right_loc)] += left_val * right_val;
    }
    __syncthreads();
#pragma unroll
    for (int ext = 0; ext < (feat_in + dim_sparse - 1) / dim_sparse; ext++)
    {
        atomicAdd(&vout[warp_row * feat_in + laneid + ext * dim_sparse], out_cache[wid * DIM_MUL(feat_in) + laneid + ext * dim_sparse]);
    }
    
}

void SPMM_OPT2_SPARSE_SHARED::run(int dim)
{
    int shared_size = (WARPS_PER_BLOCK + 0 * WARPS_PER_BLOCK / 2) * DIM_MUL(dim) * sizeof(float);

    spmm_kernel_opt2_sparse_shared<<<grid, block, shared_size>>>(_warp4, idx, val, vin, vin_sparse_selector, vout, num_v, num_e, dim, dim_sparse, num_warps);
}

double SPMM_OPT2_SPARSE_SHARED::do_test(bool timing, int dim)
{
    this->num_warps = cuda_read_array(&this->_warp4, "/home/xix22010/py_projects/graph_preprocess/warp_4/" + this->_graph + ".warp4") / 4;
    int block_num = (num_warps + (WARPS_PER_BLOCK) - 1) / (WARPS_PER_BLOCK);
    if (!timing)
    {
        cout << "block num = " << block_num << endl;
    }

    grid.x = block_num;
    block.x = WARPS_PER_BLOCK * dim_sparse;

    double ret = timing_body(timing, dim);

    hipFree(this->_warp4);
    return ret;
}