#include "hip/hip_runtime.h"
#include <iostream>
#include "data.h"
#include "spmm_ref.h"
#include "spmm_opt_sort.h"
#include "spmm_opt_sort2.h"
#include "spmm_opt_sort_innerloop.h"
#include "spmm_opt.h"
#include "spmm_opt2.h"
#include "spmm_opt_innerloop.h"
#include "spmm_gnna.h"
#include "spmm_cusparse.h"
#include "spmm_col.h"
#include <random>
#include <algorithm>
#include <filesystem>

string base_dir = "/home/xix22010/cuda_projects/hpc_data/";
// string base_dir = "/home/xiexi/PycharmProjects/pythonProject/graphs/";
// string _graph = "cora";

int total_file_cnt, current_file_cnt;

using namespace std;

double check_err(float *out, float *out_ref, int len, bool &has_err)
{
    double err_sum = 0;
    bool show = 1;

    has_err = 0;

    for (int i = 0; i < len; i++)
    {
        double err = abs(out[i] - out_ref[i]);
        err_sum += err;
        // if (err_sum / (v_num * dim) >= 0.001 && show)
        // {
        //     show = 0;
        //     cout << "fail begin at " << i/32 << endl;
        // }
        if (err > 0.1 && has_err == 0)
        {
            has_err = 1;
            cout << "err at " << i << endl;
        }
    }
    cout << "err sum = " << err_sum << "  ";
    if (err_sum / len < 0.001)
    // if(!has_err)
    {
        cout << "validation pass!" << endl;
    }
    else
    {
        cout << "validation fail!" << endl;
    }
    return err_sum;
}

void test_graph(string graph, int spec_dim)
{
    int dim_min = 160, dim_max = 512, interval = 32;
    if (spec_dim > 0)
    {
        dim_min = spec_dim;
        dim_max = spec_dim;
    }

    int *cu_indptr, *cu_indices, *cu_indptr_new, *cu_indices_new, *cu_coo_row;
    int v_num = cuda_read_array(&cu_indptr_new, base_dir + graph + ".new_indptr") - 1;
    int e_num = cuda_read_array(&cu_indices_new, base_dir + graph + ".new_indices");
    cuda_read_array(&cu_indptr, base_dir + graph + ".graph.ptrdump");
    cuda_read_array(&cu_indices, base_dir + graph + ".graph.edgedump");

    hipMallocManaged(&cu_coo_row, e_num * sizeof(int));
    {
        int k = 0;
        for (int i = 0; i < v_num; i++)
        {
            for (int j = 0; j < cu_indptr[i + 1] - cu_indptr[i]; j++)
            {
                cu_coo_row[k++] = i;
            }
        }
    }

    // cout << "graph = " << graph << " v_num = " << v_num << " e_num = " << e_num << endl;
    float *cu_val;
    hipMallocManaged(&cu_val, e_num * sizeof(float));

    float *cu_vin, *cu_vout, *cu_vout2, *cu_vout_inner, *cu_vout_new, *cu_vout_new2, *cu_vout_inner_new, *cu_vout_ref, *cu_vout_gnna, *cu_vout_ref_new, *cu_vout_ref_coo, *cu_vout_col;
    hipMallocManaged(&cu_vin, v_num * dim_max * sizeof(float));
    hipMallocManaged(&cu_vout, v_num * dim_max * sizeof(float));
    hipMallocManaged(&cu_vout2, v_num * dim_max * sizeof(float));
    hipMallocManaged(&cu_vout_inner, v_num * dim_max * sizeof(float));
    hipMallocManaged(&cu_vout_new, v_num * dim_max * sizeof(float));
    hipMallocManaged(&cu_vout_new2, v_num * dim_max * sizeof(float));
    hipMallocManaged(&cu_vout_inner_new, v_num * dim_max * sizeof(float));
    hipMallocManaged(&cu_vout_gnna, v_num * dim_max * sizeof(float));
    hipMallocManaged(&cu_vout_ref, v_num * dim_max * sizeof(float));
    hipMallocManaged(&cu_vout_ref_new, v_num * dim_max * sizeof(float));
    hipMallocManaged(&cu_vout_ref_coo, v_num * dim_max * sizeof(float));
    hipMallocManaged(&cu_vout_col, v_num * dim_max * sizeof(float));

    default_random_engine engine;
    engine.seed(123);

    uniform_real_distribution<float> rd(0, 1);

    int input_mode = 3;
    switch (input_mode)
    {
    case 1:
        generate(cu_val, cu_val + e_num, [&]()
                 { return rd(engine); });
        generate(cu_vin, cu_vin + v_num * dim_max, [&]()
                 { return rd(engine); });
        break;
    case 2:
        for (int i = 0; i < e_num; i++)
        {
            cu_val[i] = 1;
        }
        for (int i = 0; i < v_num * dim_max; i++)
        {
            cu_vin[i] = 0.01 * i;
        }
        break;
    case 3:
        for (int i = 0; i < e_num; i++)
        {
            cu_val[i] = 1;
        }
        generate(cu_vin, cu_vin + v_num * dim_max, [&]()
                 { return rd(engine); });
        break;

    default:
        break;
    }

    // fill(cu_vin, cu_vin + v_num * dim, 1);
    fill(cu_vout, cu_vout + v_num * dim_max, 0);
    fill(cu_vout_inner, cu_vout_inner + v_num * dim_max, 0);
    fill(cu_vout_gnna, cu_vout_gnna + v_num * dim_max, 0);
    fill(cu_vout_ref, cu_vout_ref + v_num * dim_max, 0);
    fill(cu_vout_ref_new, cu_vout_ref_new + v_num * dim_max, 0);

    SPMM_OPT_SORT opt_sort(graph, cu_indptr_new, cu_indices_new, cu_val, cu_vin, cu_vout_new, v_num, e_num, dim_max);
    SPMM_OPT_SORT2 opt_sort2(graph, cu_indptr_new, cu_indices_new, cu_val, cu_vin, cu_vout_new2, v_num, e_num, dim_max);
    SPMM_OPT_SORT_INNERLOOP opt_sort_innerloop(graph, cu_indptr_new, cu_indices_new, cu_val, cu_vin, cu_vout_inner_new, v_num, e_num, dim_max);
    SPMM_OPT opt(graph, cu_indptr, cu_indices, cu_val, cu_vin, cu_vout, v_num, e_num, dim_max);
    SPMM_OPT2 opt2(graph, cu_indptr, cu_indices, cu_val, cu_vin, cu_vout2, v_num, e_num, dim_max);
    SPMM_OPT_INNERLOOP opt_innerloop(graph, cu_indptr, cu_indices, cu_val, cu_vin, cu_vout_inner, v_num, e_num, dim_max);
    SPMM_GNNA gnna(graph, cu_indptr, cu_indices, cu_val, cu_vin, cu_vout_gnna, v_num, e_num, dim_max);
    SPMM_COL spmm_col(graph, cu_indptr, cu_indices, cu_val, cu_vin, cu_vout_col, v_num, e_num, dim_max);

#define CHECK
#define TIMING

    for (int dim = dim_min; dim <= dim_max; dim += interval)
    {
        // cout << "dim = " << dim << endl;

#ifdef CHECK
        spmm_cusparse(cu_indptr, cu_indices, cu_val, cu_vin, cu_vout_ref, v_num, e_num, dim, 0);
        spmm_cusparse(cu_indptr_new, cu_indices_new, cu_val, cu_vin, cu_vout_ref_new, v_num, e_num, dim, 0);
        spmm_cusparse_coo(cu_coo_row, cu_indices, cu_val, cu_vin, cu_vout_ref_coo, v_num, e_num, dim, 0);
        opt_sort.do_test(false, dim);
        opt_sort2.do_test(false, dim);
        opt_sort_innerloop.do_test(false, dim);
        opt.do_test(false, dim);
        opt2.do_test(false, dim);
        opt_innerloop.do_test(false, dim);
        gnna.do_test(false, dim);
        spmm_col.do_test(false, dim);

        bool has_err = 0;
        cout << "checking cusparse_coo" << endl;
        check_err(cu_vout_ref_coo, cu_vout_ref, v_num * dim, has_err);
        cout << "checking opt_sort" << endl;
        check_err(cu_vout_new, cu_vout_ref_new, v_num * dim, has_err);
        cout << "checking opt_sort2" << endl;
        check_err(cu_vout_new2, cu_vout_ref_new, v_num * dim, has_err);
        cout << "checking opt_sort_innerloop" << endl;
        check_err(cu_vout_inner_new, cu_vout_ref_new, v_num * dim, has_err);
        cout << "checking opt" << endl;
        check_err(cu_vout, cu_vout_ref, v_num * dim, has_err);
        cout << "checking opt2" << endl;
        check_err(cu_vout2, cu_vout_ref, v_num * dim, has_err);
        cout << "checking opt_innerloop" << endl;
        check_err(cu_vout_inner, cu_vout_ref, v_num * dim, has_err);
        cout << "checking gnna" << endl;
        check_err(cu_vout_gnna, cu_vout_ref, v_num * dim, has_err);
        cout << "checking spmm_col" << endl;
        check_err(cu_vout_col, cu_vout_ref, v_num * dim, has_err);

#endif

#ifdef TIMING
        string outstr = to_string(current_file_cnt) + "/" + to_string(total_file_cnt) + " " + graph + " " + to_string(dim);

        double t_cusparse = spmm_cusparse(cu_indptr, cu_indices, cu_val, cu_vin, cu_vout_ref, v_num, e_num, dim, 10);
        cout << outstr << " cusparse " << t_cusparse * 1000000 << endl;

        double t_cusparse_coo = spmm_cusparse_coo(cu_coo_row, cu_indices, cu_val, cu_vin, cu_vout_ref_coo, v_num, e_num, dim, 10);
        cout << outstr << " cusparse_coo " << t_cusparse_coo * 1000000 << endl;

        double t_gnna = gnna.do_test(true, dim);
        cout << outstr << " gnna " << t_gnna * 1000000 << endl;

        double t_spmm_col = spmm_col.do_test(true, dim);
        cout << outstr << " spmm_col " << t_spmm_col * 1000000 << endl;

        double t_opt_sort = opt_sort.do_test(true, dim);
        cout << outstr << " opt_sort " << t_opt_sort * 1000000 << endl;

        double t_opt_sort2 = opt_sort2.do_test(true, dim);
        cout << outstr << " opt_sort2 " << t_opt_sort2 * 1000000 << endl;

        double t_opt_sort_innerloop = opt_sort_innerloop.do_test(true, dim);
        cout << outstr << " opt_sort_innerloop " << t_opt_sort_innerloop * 1000000 << endl;

        double t_opt = opt.do_test(true, dim);
        cout << outstr << " opt " << t_opt * 1000000 << endl;

        double t_opt2 = opt2.do_test(true, dim);
        cout << outstr << " opt2 " << t_opt2 * 1000000 << endl;

        double t_opt_innerloop = opt_innerloop.do_test(true, dim);
        cout << outstr << " opt_innerloop " << t_opt_innerloop * 1000000 << endl;

#endif
    }

    hipFree(cu_indptr);
    hipFree(cu_indices);
    hipFree(cu_coo_row);
    hipFree(cu_indptr_new);
    hipFree(cu_indices_new);
    hipFree(cu_val);
    hipFree(cu_vin);
    hipFree(cu_vout);
    hipFree(cu_vout2);
    hipFree(cu_vout_inner);
    hipFree(cu_vout_new);
    hipFree(cu_vout_new2);
    hipFree(cu_vout_inner_new);
    hipFree(cu_vout_gnna);
    hipFree(cu_vout_ref);
    hipFree(cu_vout_ref_new);
    hipFree(cu_vout_ref_coo);
    hipFree(cu_vout_col);
}

int main(int argc, char *argv[])
{
    if (argc > 2)
    {
        string arg_graph(argv[1]);
        int dim = atoi(argv[2]);
        cout << "dir = " << base_dir << endl;
        test_graph(arg_graph, dim);
    }
    else
    {
        string folder_path = "/home/xix22010/cuda_projects/hpc_data/";
        string extension = ".config";

        total_file_cnt = 0;
        for (const auto &file : filesystem::directory_iterator(folder_path))
        {
            if (file.path().extension() == extension)
            {
                total_file_cnt++;
            }
        }

        current_file_cnt = 0;
        for (const auto &file : filesystem::directory_iterator(folder_path))
        {
            if (file.path().extension() == extension)
            {
                current_file_cnt++;
                if(current_file_cnt <= 31)
                    continue;

                string graph = file.path().stem().string();
                // if (!(graph == "wikikg2" || graph == "rabbit_wikikg2"))
                // {
                //     continue;
                // }
                test_graph(graph, 0);
                hipDeviceSynchronize();
            }
        }
    }

    return 0;
}