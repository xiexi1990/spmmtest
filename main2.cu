#include "hip/hip_runtime.h"
#include <iostream>
#include "data.h"
#include "spmm_ref.h"
#include "spmm_opt_sort.h"
#include "spmm_opt_sort2.h"
#include "spmm_opt_sort_innerloop.h"
#include "spmm_opt.h"
#include "spmm_opt2.h"
#include "spmm_opt2_sparse.h"
#include "spmm_opt2_sparse_shared.h"
#include "spmm_opt_innerloop.h"
#include "spmm_gnna.h"
#include "spmm_cusparse.h"
#include "spmm_col.h"
#include <random>
#include <algorithm>
#include <filesystem>

string base_dir = "/home/xix22010/cuda_projects/hpc_data/";
// string base_dir = "/home/xiexi/PycharmProjects/pythonProject/graphs/";
// string _graph = "cora";

int total_file_cnt, current_file_cnt;

using namespace std;

double check_err(float *out, float *out_ref, int len, bool &has_err)
{
    double err_sum = 0;
    bool show = 1;

    has_err = 0;

    for (int i = 0; i < len; i++)
    {
        double err = abs(out[i] - out_ref[i]);
        err_sum += err;
        // if (err_sum / (v_num * dim) >= 0.001 && show)
        // {
        //     show = 0;
        //     cout << "fail begin at " << i/32 << endl;
        // }
        if (err > 0.1 && has_err == 0)
        {
            has_err = 1;
            cout << "err at " << i << endl;
        }
    }
    cout << "err sum = " << err_sum << "  ";
    if (err_sum / len < 0.001)
    // if(!has_err)
    {
        cout << "validation pass!" << endl;
    }
    else
    {
        cout << "validation fail!" << endl;
    }
    return err_sum;
}

void test_graph(string graph, int spec_dim)
{
    int dim_min = 256, dim_max = 256, interval = 1;
    int dim_sparse = 16;
    cout << "dim sparse = " << dim_sparse << endl;

    if (spec_dim > 0)
    {
        dim_min = spec_dim;
        dim_max = spec_dim;
    }

    int *cu_indptr, *cu_indices, *cu_indptr_new, *cu_indices_new, *cu_coo_row;
    int v_num = cuda_read_array(&cu_indptr_new, base_dir + graph + ".new_indptr") - 1;
    int e_num = cuda_read_array(&cu_indices_new, base_dir + graph + ".new_indices");
    cuda_read_array(&cu_indptr, base_dir + graph + ".graph.ptrdump");
    cuda_read_array(&cu_indices, base_dir + graph + ".graph.edgedump");

    hipMallocManaged(&cu_coo_row, e_num * sizeof(int));
    {
        int k = 0;
        for (int i = 0; i < v_num; i++)
        {
            for (int j = 0; j < cu_indptr[i + 1] - cu_indptr[i]; j++)
            {
                cu_coo_row[k++] = i;
            }
        }
    }

    // cout << "graph = " << graph << " v_num = " << v_num << " e_num = " << e_num << endl;
    float *cu_val;
    hipMallocManaged(&cu_val, e_num * sizeof(float));

    float *cu_vin, *cu_vout, *cu_vout2, *cu_vout_inner, *cu_vout_new, *cu_vout_new2, *cu_vout_inner_new, *cu_vout_ref, *cu_vout_gnna, *cu_vout_ref_new, *cu_vout_ref_coo, *cu_vout_col;
    float *cu_vin_sparse, *cu_vin_sparse_data, *cu_vout2_sparse, *cu_vout2_sparse_shared;
    int *cu_vin_sparse_selector;
    hipMallocManaged(&cu_vin, v_num * dim_max * sizeof(float));
    hipMallocManaged(&cu_vout, v_num * dim_max * sizeof(float));
    hipMallocManaged(&cu_vout2, v_num * dim_max * sizeof(float));
    hipMallocManaged(&cu_vout_inner, v_num * dim_max * sizeof(float));
    hipMallocManaged(&cu_vout_new, v_num * dim_max * sizeof(float));
    hipMallocManaged(&cu_vout_new2, v_num * dim_max * sizeof(float));
    hipMallocManaged(&cu_vout_inner_new, v_num * dim_max * sizeof(float));
    hipMallocManaged(&cu_vout_gnna, v_num * dim_max * sizeof(float));
    hipMallocManaged(&cu_vout_ref, v_num * dim_max * sizeof(float));
    hipMallocManaged(&cu_vout_ref_new, v_num * dim_max * sizeof(float));
    hipMallocManaged(&cu_vout_ref_coo, v_num * dim_max * sizeof(float));
    hipMallocManaged(&cu_vout_col, v_num * dim_max * sizeof(float));

    hipMallocManaged(&cu_vin_sparse, v_num * dim_max * sizeof(float));
    hipMallocManaged(&cu_vin_sparse_data, v_num * dim_sparse * sizeof(float));
    hipMallocManaged(&cu_vin_sparse_selector, v_num * dim_sparse * sizeof(int));
    hipMallocManaged(&cu_vout2_sparse, v_num * dim_max * sizeof(float));
    hipMallocManaged(&cu_vout2_sparse_shared, v_num * dim_max * sizeof(float));


    default_random_engine engine;
    engine.seed(123);

    uniform_real_distribution<float> rd(0, 1);

    int input_mode = 3;
    switch (input_mode)
    {
    case 1:
        generate(cu_val, cu_val + e_num, [&]()
                 { return rd(engine); });
        generate(cu_vin, cu_vin + v_num * dim_max, [&]()
                 { return rd(engine); });
        break;
    case 2:
        for (int i = 0; i < e_num; i++)
        {
            cu_val[i] = 1;
        }
        for (int i = 0; i < v_num * dim_max; i++)
        {
            cu_vin[i] = 0.01 * i;
        }
        break;
    case 3:
        for (int i = 0; i < e_num; i++)
        {
            cu_val[i] = 1;
        }
        generate(cu_vin, cu_vin + v_num * dim_max, [&]()
                 { return rd(engine); });
        break;

    default:
        break;
    }

    vector<int> sequence(dim_max);
    iota(sequence.begin(), sequence.end(), 0);  // Fill with 0, 1, ..., dim_max - 1

    vector<int> sample(dim_sparse);

    for (int i = 0; i < v_num; ++i) {
        std::sample(sequence.begin(), sequence.end(), sample.begin(), dim_sparse, engine);

        for (int j = 0; j < dim_sparse; ++j) {
            cu_vin_sparse_data[i * dim_sparse + j] = rd(engine);
            cu_vin_sparse_selector[i * dim_sparse + j] = sample[j];
        }
    }

    for (int i = 0; i < v_num; ++i) {
        for (int j = 0; j < dim_max; ++j) {
            cu_vin_sparse[i * dim_max + j] = 0.0;
        }
        for (int j = 0; j < dim_sparse; ++j) {
            int col = cu_vin_sparse_selector[i * dim_sparse + j];
            cu_vin_sparse[i * dim_max + col] = cu_vin_sparse_data[i * dim_sparse + j];
        }
    }

    // for(int i = 0; i < 100; i++){
    //     cout<<cu_vin_sparse_data[i]<<" ";
    // }
    // cout<<endl<<endl;
    // for(int i = 0; i < 100; i++){
    //     cout<<cu_vin_sparse_selector[i]<<" ";
    // }
    // cout<<endl<<endl;
    // for(int i = 0; i < 100; i++){
    //     cout<<cu_vin_sparse[i]<<" ";
    // }
    // cout<<endl<<endl;


    // fill(cu_vin, cu_vin + v_num * dim, 1);
    fill(cu_vout, cu_vout + v_num * dim_max, 0);
    fill(cu_vout_inner, cu_vout_inner + v_num * dim_max, 0);
    fill(cu_vout_gnna, cu_vout_gnna + v_num * dim_max, 0);
    fill(cu_vout_ref, cu_vout_ref + v_num * dim_max, 0);
    fill(cu_vout_ref_new, cu_vout_ref_new + v_num * dim_max, 0);

    fill(cu_vout2, cu_vout2 + v_num * dim_max, 0);
    fill(cu_vout2_sparse, cu_vout2_sparse + v_num * dim_max, 0);

    // SPMM_OPT_SORT opt_sort(graph, cu_indptr_new, cu_indices_new, cu_val, cu_vin, cu_vout_new, v_num, e_num, dim_max);
    // SPMM_OPT_SORT2 opt_sort2(graph, cu_indptr_new, cu_indices_new, cu_val, cu_vin, cu_vout_new2, v_num, e_num, dim_max);
    // SPMM_OPT_SORT_INNERLOOP opt_sort_innerloop(graph, cu_indptr_new, cu_indices_new, cu_val, cu_vin, cu_vout_inner_new, v_num, e_num, dim_max);
    // SPMM_OPT opt(graph, cu_indptr, cu_indices, cu_val, cu_vin, cu_vout, v_num, e_num, dim_max);

    SPMM_OPT2 opt2(graph, cu_indptr, cu_indices, cu_val, cu_vin_sparse, cu_vout2, v_num, e_num, dim_max);

    // SPMM_OPT_INNERLOOP opt_innerloop(graph, cu_indptr, cu_indices, cu_val, cu_vin, cu_vout_inner, v_num, e_num, dim_max);
    // SPMM_GNNA gnna(graph, cu_indptr, cu_indices, cu_val, cu_vin, cu_vout_gnna, v_num, e_num, dim_max);
    // SPMM_COL spmm_col(graph, cu_indptr, cu_indices, cu_val, cu_vin, cu_vout_col, v_num, e_num, dim_max);

    SPMM_OPT2_SPARSE opt2_sparse(graph, cu_indptr, cu_indices, cu_val, cu_vin_sparse_data, cu_vout2_sparse, v_num, e_num, dim_max);
    opt2_sparse.vin_sparse_selector = cu_vin_sparse_selector;
    opt2_sparse.dim_sparse = dim_sparse;

    SPMM_OPT2_SPARSE_SHARED opt2_sparse_shared(graph, cu_indptr, cu_indices, cu_val, cu_vin_sparse_data, cu_vout2_sparse_shared, v_num, e_num, dim_max);
    opt2_sparse_shared.vin_sparse_selector = cu_vin_sparse_selector;
    opt2_sparse_shared.dim_sparse = dim_sparse;

#define CHECK
#define TIMING

    for (int dim = dim_min; dim <= dim_max; dim += interval)
    {
        // cout << "dim = " << dim << endl;

#ifdef CHECK
        spmm_cusparse(cu_indptr, cu_indices, cu_val, cu_vin_sparse, cu_vout_ref, v_num, e_num, dim, 0);

        // for(int i = 0; i < 300; i++){
        //     cout<<cu_vout_ref[i]<<" ";
        // }
        // cout<<endl<<endl;

        // spmm_cusparse(cu_indptr_new, cu_indices_new, cu_val, cu_vin, cu_vout_ref_new, v_num, e_num, dim, 0);
        // spmm_cusparse_coo(cu_coo_row, cu_indices, cu_val, cu_vin, cu_vout_ref_coo, v_num, e_num, dim, 0);
        // opt_sort.do_test(false, dim);
        // opt_sort2.do_test(false, dim);
        // opt_sort_innerloop.do_test(false, dim);
        // opt.do_test(false, dim);
        opt2.do_test(false, dim);

        // for(int i = 0; i < 300; i++){
        //     cout<<cu_vout2[i]<<" ";
        // }
        // cout<<endl<<endl;

        opt2_sparse.do_test(false, dim);

        opt2_sparse_shared.do_test(false, dim);

        // for(int i = 0; i < 300; i++){
        //     cout<<cu_vout2_sparse[i]<<" ";
        // }
        // cout<<endl<<endl;

        // opt_innerloop.do_test(false, dim);
        // gnna.do_test(false, dim);
        // spmm_col.do_test(false, dim);

        bool has_err = 0;
        // cout << "checking cusparse_coo" << endl;
        // check_err(cu_vout_ref_coo, cu_vout_ref, v_num * dim, has_err);
        // cout << "checking opt_sort" << endl;
        // check_err(cu_vout_new, cu_vout_ref_new, v_num * dim, has_err);
        // cout << "checking opt_sort2" << endl;
        // check_err(cu_vout_new2, cu_vout_ref_new, v_num * dim, has_err);
        // cout << "checking opt_sort_innerloop" << endl;
        // check_err(cu_vout_inner_new, cu_vout_ref_new, v_num * dim, has_err);
        // cout << "checking opt" << endl;
        // check_err(cu_vout, cu_vout_ref, v_num * dim, has_err);
        cout << "checking opt2" << endl;
        check_err(cu_vout2, cu_vout_ref, v_num * dim, has_err);

        cout << "checking opt2_sparse" << endl;
        check_err(cu_vout2_sparse, cu_vout_ref, v_num * dim, has_err);

        cout << "checking opt2_sparse_shared" << endl;
        check_err(cu_vout2_sparse_shared, cu_vout_ref, v_num * dim, has_err);

        // cout << "checking opt_innerloop" << endl;
        // check_err(cu_vout_inner, cu_vout_ref, v_num * dim, has_err);
        // cout << "checking gnna" << endl;
        // check_err(cu_vout_gnna, cu_vout_ref, v_num * dim, has_err);
        // cout << "checking spmm_col" << endl;
        // check_err(cu_vout_col, cu_vout_ref, v_num * dim, has_err);

#endif

#ifdef TIMING
        string outstr = to_string(current_file_cnt) + "/" + to_string(total_file_cnt) + " " + graph + " " + to_string(dim);

        double t_cusparse = spmm_cusparse(cu_indptr, cu_indices, cu_val, cu_vin, cu_vout_ref, v_num, e_num, dim, 10);
        cout << outstr << " cusparse " << t_cusparse * 1000 << endl;

        // double t_cusparse_coo = spmm_cusparse_coo(cu_coo_row, cu_indices, cu_val, cu_vin, cu_vout_ref_coo, v_num, e_num, dim, 10);
        // cout << outstr << " cusparse_coo " << t_cusparse_coo * 1000000 << endl;

        // double t_gnna = gnna.do_test(true, dim);
        // cout << outstr << " gnna " << t_gnna * 1000000 << endl;

        // double t_spmm_col = spmm_col.do_test(true, dim);
        // cout << outstr << " spmm_col " << t_spmm_col * 1000000 << endl;

        // double t_opt_sort = opt_sort.do_test(true, dim);
        // cout << outstr << " opt_sort " << t_opt_sort * 1000000 << endl;

        // double t_opt_sort2 = opt_sort2.do_test(true, dim);
        // cout << outstr << " opt_sort2 " << t_opt_sort2 * 1000000 << endl;

        // double t_opt_sort_innerloop = opt_sort_innerloop.do_test(true, dim);
        // cout << outstr << " opt_sort_innerloop " << t_opt_sort_innerloop * 1000000 << endl;

        // double t_opt = opt.do_test(true, dim);
        // cout << outstr << " opt " << t_opt * 1000000 << endl;

        double t_opt2 = opt2.do_test(true, dim);
        cout << outstr << " opt2 " << t_opt2 * 1000 << endl;

        double t_opt2_sparse = opt2_sparse.do_test(true, dim);
        cout << outstr << " opt2_sparse " << t_opt2_sparse * 1000 << endl;

        double t_opt2_sparse_shared = opt2_sparse_shared.do_test(true, dim);
        cout << outstr << " opt2_sparse_shared " << t_opt2_sparse_shared * 1000 << endl;

        // double t_opt_innerloop = opt_innerloop.do_test(true, dim);
        // cout << outstr << " opt_innerloop " << t_opt_innerloop * 1000000 << endl;

#endif
    }

    hipFree(cu_indptr);
    hipFree(cu_indices);
    hipFree(cu_coo_row);
    hipFree(cu_indptr_new);
    hipFree(cu_indices_new);
    hipFree(cu_val);
    hipFree(cu_vin);
    hipFree(cu_vout);
    hipFree(cu_vout2);
    hipFree(cu_vout_inner);
    hipFree(cu_vout_new);
    hipFree(cu_vout_new2);
    hipFree(cu_vout_inner_new);
    hipFree(cu_vout_gnna);
    hipFree(cu_vout_ref);
    hipFree(cu_vout_ref_new);
    hipFree(cu_vout_ref_coo);
    hipFree(cu_vout_col);

    hipFree(cu_vin_sparse);
    hipFree(cu_vin_sparse_data);
    hipFree(cu_vin_sparse_selector);
    hipFree(cu_vout2_sparse);
    hipFree(cu_vout2_sparse_shared);
}

int main(int argc, char *argv[])
{
    if (argc > 2)
    {
        string arg_graph(argv[1]);
        int dim = atoi(argv[2]);
        cout << "dir = " << base_dir << endl;
        test_graph(arg_graph, dim);
    }
    else
    {
        string folder_path = "/home/xix22010/cuda_projects/hpc_data/";
        string extension = ".config";

        total_file_cnt = 0;
        for (const auto &file : filesystem::directory_iterator(folder_path))
        {
            if (file.path().extension() == extension)
            {
                total_file_cnt++;
            }
        }

        current_file_cnt = 0;
        for (const auto &file : filesystem::directory_iterator(folder_path))
        {
            if (file.path().extension() == extension)
            {
                current_file_cnt++;

                string graph = file.path().stem().string();
                // if (!(graph == "wikikg2" || graph == "rabbit_wikikg2"))
                // {
                //     continue;
                // }
                test_graph(graph, 0);
                hipDeviceSynchronize();
            }
        }
    }

    return 0;
}