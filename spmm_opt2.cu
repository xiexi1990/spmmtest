#include "hip/hip_runtime.h"
#include "spmm_opt2.h"
#include "data.h"
#include <string>
#include <iostream>
#define CONSTINT const int

using namespace std;

extern string base_dir, graph;

const int WARPS_PER_BLOCK = 12;

__global__ void spmm_kernel_opt2(const int *_warp4, const int *idx, const float *val, const float *vin, float *vout, const int num_v, const int num_e, const int feat_in, const int num_warps)
{
    const int4 *warp4 = reinterpret_cast<const int4 *>(_warp4);
    // extern __shared__ float out_cache[];

    CONSTINT dim_mul = (feat_in + 31) / 32;
    CONSTINT round_dim = dim_mul * 32;

#pragma unroll
    for (int ext = 0; ext < dim_mul; ext++)
    {
        const int tid = blockIdx.x * blockDim.x * dim_mul + threadIdx.x + ext * blockDim.x; 
        const int warpid = tid / round_dim;                           
        const int block_warpid = threadIdx.x / round_dim;            
        const int laneid = threadIdx.x % round_dim;   
        if (warpid >= num_warps || laneid >= feat_in)
            return; 
        const int4 w_info = warp4[warpid];
        CONSTINT warp_row = w_info.x;
        CONSTINT warp_loc = w_info.y;
        CONSTINT warp_len = w_info.z;

        float tmp = 0;
#pragma unroll
        for (int i = 0; i < warp_len; i++)
        {
            const int nz_loc = warp_loc + i;
            const float left_val = __ldg(val + nz_loc);
            const float right_val = vin[__ldg(idx + nz_loc) * feat_in + laneid];
            tmp += left_val * right_val;
        }
        atomicAdd(&vout[warp_row * feat_in + laneid], tmp);
    }
}

void SPMM_OPT2::run(int dim)
{
    spmm_kernel_opt2<<<grid, block>>>(_warp4, idx, val, vin, vout, num_v, num_e, dim, num_warps);
}

double SPMM_OPT2::do_test(bool timing, int dim)
{
    this->num_warps = cuda_read_array(&this->_warp4, "/home/xix22010/py_projects/graph_preprocess/warp_4/" + this->_graph + ".warp4") / 4;
    int block_num = (num_warps + WARPS_PER_BLOCK - 1) / WARPS_PER_BLOCK;
    if (!timing)
    {
        cout << "block num = " << block_num << endl;
    }

    grid.x = block_num;
    block.x = WARPS_PER_BLOCK * 32;

    double ret = timing_body(timing, dim);

    hipFree(this->_warp4);
    return ret;
}