#include "hip/hip_runtime.h"
#include <iostream>
#include "data.h"
#include "spmm_opt2.h"
#include "spmm_opt2_sparse.h"
#include "spmm_opt2_sparse_shared.h"
#include "spmm_opt2_sparse_v3.h"
#include "spmm_opt2_sparse_backward_v3.h"
#include "spmm_cusparse.h"
#include <random>
#include <algorithm>
#include <filesystem>

string base_dir = "/home/xix22010/cuda_projects/hpc_data/";
// string base_dir = "/home/xiexi/PycharmProjects/pythonProject/graphs/";
// string _graph = "cora";

int total_file_cnt, current_file_cnt;

using namespace std;

#define DIM_MUL_N 1
#define DIM_MUL(x) ((x + DIM_MUL_N - 1) / DIM_MUL_N) * DIM_MUL_N

double check_err(float *out, float *out_ref, int len, bool &has_err)
{
    double err_sum = 0;
    bool show = 1;

    has_err = 0;

    for (int i = 0; i < len; i++)
    {
        double err = abs(out[i] - out_ref[i]);
        err_sum += err;
        // if (err_sum / (v_num * dim) >= 0.001 && show)
        // {
        //     show = 0;
        //     cout << "fail begin at " << i/32 << endl;
        // }
        if (err > 0.1 && has_err == 0)
        {
            has_err = 1;
            cout << "err at " << i << endl;
        }
    }
    cout << "err sum = " << err_sum << "  ";
    if (err_sum / len < 0.001)
    // if(!has_err)
    {
        cout << "validation pass!" << endl;
    }
    else
    {
        cout << "validation fail!" << endl;
    }
    return err_sum;
}

void test_graph(string graph, int spec_dim, int dim_sparse)
{
    int dim_min = 256, dim_max = 256, interval = 1;

    cout << "dim sparse = " << dim_sparse << endl;

    if (spec_dim > 0)
    {
        dim_min = spec_dim;
        dim_max = spec_dim;
    }

    int *cu_indptr, *cu_indices;
    int v_num = cuda_read_array(&cu_indptr, base_dir + graph + ".graph.ptrdump") - 1;
    int e_num = cuda_read_array(&cu_indices, base_dir + graph + ".graph.edgedump");

    // cout << "graph = " << graph << " v_num = " << v_num << " e_num = " << e_num << endl;
    float *cu_val;
    hipMallocManaged(&cu_val, e_num * sizeof(float));

    float *cu_vout2, *cu_vout_ref, *cu_vout_ref_to_backward;
    float *cu_vin_sparse, *cu_vin_sparse_data, *cu_vout2_sparse, *cu_vout2_sparse_shared, *cu_vout2_sparse_v3, *cu_vout2_sparse_backward;
    int *cu_vin_sparse_selector;
    hipMallocManaged(&cu_vout2, v_num * dim_max * sizeof(float));
    hipMallocManaged(&cu_vout_ref, v_num * dim_max * sizeof(float));
    hipMallocManaged(&cu_vin_sparse, v_num * dim_max * sizeof(float));
    hipMallocManaged(&cu_vin_sparse_data, v_num * DIM_MUL(dim_sparse) * sizeof(float));
    hipMallocManaged(&cu_vout_ref_to_backward, v_num * DIM_MUL(dim_sparse) * sizeof(float));
    hipMallocManaged(&cu_vout2_sparse_backward, v_num * DIM_MUL(dim_sparse) * sizeof(float));

    hipMallocManaged(&cu_vin_sparse_selector, v_num * DIM_MUL(dim_sparse) * sizeof(int));

    hipMallocManaged(&cu_vout2_sparse, v_num * dim_max * sizeof(float));
    hipMallocManaged(&cu_vout2_sparse_shared, v_num * dim_max * sizeof(float));
    hipMallocManaged(&cu_vout2_sparse_v3, v_num * dim_max * sizeof(float));
    

    default_random_engine engine;
    engine.seed(123);

    uniform_real_distribution<float> rd(0, 1);

    int input_mode = 3;
    switch (input_mode)
    {
    case 1:
        generate(cu_val, cu_val + e_num, [&]()
                 { return rd(engine); });
        // generate(cu_vin, cu_vin + v_num * dim_max, [&]()
        //          { return rd(engine); });
        break;
    case 2:
        for (int i = 0; i < e_num; i++)
        {
            cu_val[i] = 1;
        }
        // for (int i = 0; i < v_num * dim_max; i++)
        // {
        //     cu_vin[i] = 0.01 * i;
        // }
        break;
    case 3:
        for (int i = 0; i < e_num; i++)
        {
            cu_val[i] = 1;
        }
        // generate(cu_vin, cu_vin + v_num * dim_max, [&]()
        //          { return rd(engine); });
        break;

    default:
        break;
    }

    vector<int> sequence(dim_max);
    iota(sequence.begin(), sequence.end(), 0); // Fill with 0, 1, ..., dim_max - 1

    vector<int> sample(dim_sparse);

    int cnt = 1;
    for (int i = 0; i < v_num; ++i)
    {
        std::sample(sequence.begin(), sequence.end(), sample.begin(), dim_sparse, engine);

        for (int j = 0; j < dim_sparse; ++j)
        {
            float v = rd(engine);
            // float v = cnt++ * 0.01;
            cu_vin_sparse_data[i * DIM_MUL(dim_sparse) + j] = v;
            cu_vin_sparse_selector[i * DIM_MUL(dim_sparse) + j] = sample[j];
        }
    }

    for (int i = 0; i < v_num; ++i)
    {
        for (int j = 0; j < dim_max; ++j)
        {
            cu_vin_sparse[i * dim_max + j] = 0.0;
        }
        for (int j = 0; j < dim_sparse; ++j)
        {
            int col = cu_vin_sparse_selector[i * DIM_MUL(dim_sparse) + j];
            cu_vin_sparse[i * dim_max + col] = cu_vin_sparse_data[i * DIM_MUL(dim_sparse) + j];
        }
    }

    // for(int i = 0; i < 200; i++){
    //     cout<<cu_vin_sparse_data[i]<<" ";
    // }
    // cout<<endl<<endl;

    // for(int i = 0; i < 200; i++){
    //     cout<<cu_vin_sparse_selector[i]<<" ";
    // }
    // cout<<endl<<endl;

    // for(int i = 0; i < 200; i++){
    //     cout<<cu_vin_sparse[i]<<" ";
    // }
    // cout<<endl<<endl;

    fill(cu_vout2, cu_vout2 + v_num * dim_max, 0);
    fill(cu_vout_ref, cu_vout_ref + v_num * dim_max, 0);
    fill(cu_vout2_sparse, cu_vout2_sparse + v_num * dim_max, 0);
    fill(cu_vout2_sparse_shared, cu_vout2_sparse_shared + v_num * dim_max, 0);
    fill(cu_vout2_sparse_v3, cu_vout2_sparse_v3 + v_num * dim_max, 0);
    fill(cu_vout2_sparse_backward, cu_vout2_sparse_backward + v_num * dim_sparse, 0);

    SPMM_OPT2 opt2(graph, cu_indptr, cu_indices, cu_val, cu_vin_sparse, cu_vout2, v_num, e_num, dim_max);

    SPMM_OPT2_SPARSE opt2_sparse(graph, cu_indptr, cu_indices, cu_val, cu_vin_sparse_data, cu_vout2_sparse, v_num, e_num, dim_max);
    opt2_sparse.vin_sparse_selector = cu_vin_sparse_selector;
    opt2_sparse.dim_sparse = dim_sparse;

    SPMM_OPT2_SPARSE_SHARED opt2_sparse_shared(graph, cu_indptr, cu_indices, cu_val, cu_vin_sparse_data, cu_vout2_sparse_shared, v_num, e_num, dim_max);
    opt2_sparse_shared.vin_sparse_selector = cu_vin_sparse_selector;
    opt2_sparse_shared.dim_sparse = dim_sparse;

    SPMM_OPT2_SPARSE_V3 opt2_sparse_v3(graph, cu_indptr, cu_indices, cu_val, cu_vin_sparse_data, cu_vout2_sparse_v3, v_num, e_num, dim_max);
    opt2_sparse_v3.vin_sparse_selector = cu_vin_sparse_selector;
    opt2_sparse_v3.dim_sparse = dim_sparse;

    SPMM_OPT2_SPARSE_BACKWARD_V3 opt2_sparse_backward_v3(graph, cu_indptr, cu_indices, cu_val, cu_vin_sparse, cu_vout2_sparse_backward, v_num, e_num, dim_max);
    opt2_sparse_backward_v3.vin_sparse_selector = cu_vin_sparse_selector;
    opt2_sparse_backward_v3.dim_sparse = dim_sparse;

    // for(int i = 0; i < 100; i++){
    //         cout << cu_vin_sparse_data[i] << " ";
    //     }
    //     cout<<endl<<endl;
    //     for(int i = 0; i < 100; i++){
    //         cout << cu_vin_sparse_selector[i] << " ";
    //     }
    //     cout<<endl<<endl;

#define CHECK
#define TIMING

    for (int dim = dim_min; dim <= dim_max; dim += interval)
    {
        // cout << "dim = " << dim << endl;

#ifdef CHECK
        spmm_cusparse(cu_indptr, cu_indices, cu_val, cu_vin_sparse, cu_vout_ref, v_num, e_num, dim, 0);

        for(int i = 0; i < v_num; i++){
            for(int j = 0; j < dim_sparse; j++){
                cu_vout_ref_to_backward[i * dim_sparse + j] = cu_vout_ref[i * dim_max + cu_vin_sparse_selector[i * dim_sparse + j]];
            }
        }


        // for(int i = 0; i < 100; i++){
        //     cout << cu_vout_ref_to_backward[i] << " ";
        // }
        // cout << endl << endl;

        // opt2.do_test(false, dim);
        // opt2_sparse.do_test(false, dim);
        // opt2_sparse_shared.do_test(false, dim);
        opt2_sparse_v3.do_test(false, dim);

        opt2_sparse_backward_v3.do_test(false, dim);
        
        // for(int i = 0; i < 100; i++){
        //     cout << cu_vout2_sparse_backward[i] << " ";
        // }
        // cout << endl << endl;

        // for(int i = 0; i < 100; i++){
        //     cout << cu_vout2_sparse_backward[i] - cu_vout_ref_to_backward[i] << " ";
        // }
        // cout << endl << endl;


        bool has_err = 0;

        // cout << "checking opt2" << endl;
        // check_err(cu_vout2, cu_vout_ref, v_num * dim, has_err);

        // for(int i = 0; i < 600; i++){
        //     cout << cu_vout2[i] << " ";
        // }
        // cout << endl << endl;

        // cout << "checking opt2_sparse" << endl;
        // check_err(cu_vout2_sparse, cu_vout_ref, v_num * dim, has_err);

        // cout << "checking opt2_sparse_shared" << endl;
        // check_err(cu_vout2_sparse_shared, cu_vout_ref, v_num * dim, has_err);

        cout << "checking opt2_sparse_v3" << endl;
        check_err(cu_vout2_sparse_v3, cu_vout_ref, v_num * dim, has_err);

        cout << "checking opt2_sparse_backward_v3" << endl;
        check_err(cu_vout2_sparse_backward, cu_vout_ref_to_backward, v_num * dim_sparse, has_err);

        // for(int i = 0; i < 300; i++){
        //     cout << cu_vout2_sparse_v3[i] << " ";
        // }
        // cout << endl << endl;

#endif

#ifdef TIMING
        string outstr = to_string(current_file_cnt) + "/" + to_string(total_file_cnt) + " " + graph + " " + to_string(dim);

        double t_cusparse = spmm_cusparse(cu_indptr, cu_indices, cu_val, cu_vin_sparse, cu_vout_ref, v_num, e_num, dim, 10);
        cout << outstr << " cusparse " << t_cusparse * 1000 << endl;

        // double t_opt2 = opt2.do_test(true, dim);
        // cout << outstr << " opt2 " << t_opt2 * 1000 << endl;

        // double t_opt2_sparse = opt2_sparse.do_test(true, dim);
        // cout << outstr << " opt2_sparse " << t_opt2_sparse * 1000 << endl;

        // double t_opt2_sparse_shared = opt2_sparse_shared.do_test(true, dim);
        // cout << outstr << " opt2_sparse_shared " << t_opt2_sparse_shared * 1000 << endl;

        double t_opt2_sparse_v3 = opt2_sparse_v3.do_test(true, dim);
        cout << outstr << " opt2_sparse_v3 " << t_opt2_sparse_v3 * 1000 << endl;

        double t_opt2_sparse_backward_v3 = opt2_sparse_backward_v3.do_test(true, dim);
        cout << outstr << " opt2_sparse_backward_v3 " << t_opt2_sparse_backward_v3 * 1000 << endl;

#endif
    }

    hipFree(cu_indptr);
    hipFree(cu_indices);
    hipFree(cu_val);
    hipFree(cu_vout2);
    hipFree(cu_vout_ref);
    hipFree(cu_vout_ref_to_backward);

    hipFree(cu_vin_sparse);
    hipFree(cu_vin_sparse_data);
    hipFree(cu_vin_sparse_selector);
    hipFree(cu_vout2_sparse);
    hipFree(cu_vout2_sparse_shared);
    hipFree(cu_vout2_sparse_v3);
    hipFree(cu_vout2_sparse_backward);
}

int main(int argc, char *argv[])
{
    if (argc > 3)
    {
        string arg_graph(argv[1]);
        int dim = atoi(argv[2]);
        int dim_sparse = atoi(argv[3]);
        cout << "dir = " << base_dir << endl;
        test_graph(arg_graph, dim, dim_sparse);
    }
    else
    {
        string folder_path = "/home/xix22010/cuda_projects/hpc_data/";
        string extension = ".config";

        total_file_cnt = 0;
        for (const auto &file : filesystem::directory_iterator(folder_path))
        {
            if (file.path().extension() == extension)
            {
                total_file_cnt++;
            }
        }

        current_file_cnt = 0;
        for (const auto &file : filesystem::directory_iterator(folder_path))
        {
            if (file.path().extension() == extension)
            {
                current_file_cnt++;

                string graph = file.path().stem().string();
                // if (!(graph == "wikikg2" || graph == "rabbit_wikikg2"))
                // {
                //     continue;
                // }
                test_graph(graph, 0, 32);
                hipDeviceSynchronize();
            }
        }
    }

    return 0;
}