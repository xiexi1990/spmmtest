#include "hip/hip_runtime.h"
#include "spmm_opt_sort_innerloop.h"
#include "data.h"
#include <string>
#include <iostream>
#define CONSTINT const int

using namespace std;

extern string base_dir, graph;

const int DEG_BOUND = 12 * 32;
const int WARPS_PER_BLOCK = 12;

#define DIM_MUL(x) ((x + 31) / 32) * 32

__global__ void spmm_kernel_opt_sort_innerloop(const int *_block4, const int *coo_row, const int *idx, const float *val, const float *vin, float *vout, const int num_v, const int num_e, const int feat_in, const float *vout_ref)
{
    const int4 *block4 = reinterpret_cast<const int4 *>(_block4);
    const int4 b_info = block4[blockIdx.x];

    CONSTINT block_degree = b_info.x;
    CONSTINT block_row_begin = b_info.y;
    CONSTINT block_loc_begin = b_info.z;
    CONSTINT block_info = b_info.w;

    CONSTINT n_rows = block_degree <= DEG_BOUND ? block_info & 65535 : 1;
    CONSTINT w_nz = block_degree <= DEG_BOUND ? block_info >> 16 : DEG_BOUND / WARPS_PER_BLOCK;
    CONSTINT row_nz = block_degree <= DEG_BOUND ? block_degree : block_info;

    extern __shared__ float out_cache[];

    CONSTINT round_dim = DIM_MUL(feat_in);

    CONSTINT warps_per_row = (row_nz + w_nz - 1) / w_nz;

    CONSTINT lane_id = threadIdx.x % 32;

    CONSTINT wid = threadIdx.x / 32;

    CONSTINT warp_loc_row = wid / warps_per_row;
    CONSTINT warp_loc_col = wid % warps_per_row * w_nz;

    if (warp_loc_row >= n_rows)
    {
        return;
    }

#pragma unroll
    for (int i = 0; i < w_nz; i++)
    {
        if (i + warp_loc_col >= row_nz)
        {
            break;
        }
        if (i == 0)
        {
            for (int d = lane_id; d < feat_in; d += 32)
            {
                out_cache[wid * round_dim + d] = 0;
#ifdef BLOCK_SUMUP
                if (warps_per_row > 1 && wid < n_rows)
                {
                    out_cache[(wid + WARPS_PER_BLOCK) * round_dim + d] = 0;
                }
#endif
            }
#ifdef BLOCK_SUMUP
            __syncwarp();
#endif
        }
        const int nz_loc = block_loc_begin + warp_loc_row * row_nz + i + warp_loc_col;
        const float left_val = __ldg(val + nz_loc);

        for (int d = lane_id; d < feat_in; d += 32)
        {
            float right_val = vin[__ldg(idx + nz_loc) * feat_in + d];
            out_cache[wid * round_dim + d] += left_val * right_val;
        }
        // out_cache[wid * feat_in + lane_id + j * 32] += right_val;
    }

    // atomicAdd(&vout[(block_row_begin + warp_loc_row) * feat_in + lane_id], out_cache[wid * round_dim + lane_id]);

    if (warps_per_row > 1)
    {

#ifdef BLOCK_SUMUP
        for (int d = lane_id; d < feat_in; d += 32)
        {
            atomicAdd_block(&out_cache[(warp_loc_row + WARPS_PER_BLOCK) * round_dim + d], out_cache[wid * round_dim + d]);
        }

        __syncthreads();
        if (wid < n_rows)
        {

            // if(vout[(block_row_begin + wid) * feat_in + lane_id] - vout_ref[(block_row_begin + wid) * feat_in + lane_id] > 0.01){
            //     ;
            // }

            if (block_degree <= DEG_BOUND)
            {
                for (int d = lane_id; d < feat_in; d += 32)
                {
                    vout[(block_row_begin + wid) * feat_in + d] = out_cache[(wid + WARPS_PER_BLOCK) * round_dim + d];
                }
            }
            else
            {
                for (int d = lane_id; d < feat_in; d += 32)
                {
                    atomicAdd(&vout[(block_row_begin + wid) * feat_in + d], out_cache[(wid + WARPS_PER_BLOCK) * round_dim + d]);
                }
            }
        }
#else
        for (int d = lane_id; d < feat_in; d += 32)
        {
            atomicAdd(&vout[(block_row_begin + warp_loc_row) * feat_in + d], out_cache[wid * round_dim + d]);
        }
#endif
    }
    else
    {
        if (block_degree <= DEG_BOUND)
        {
            for (int d = lane_id; d < feat_in; d += 32)
            {
                vout[(block_row_begin + wid) * feat_in + d] = out_cache[wid * round_dim + d];
            }
        }

        else
        {
            for (int d = lane_id; d < feat_in; d += 32)
            {
                atomicAdd(&vout[(block_row_begin + wid) * feat_in + d], out_cache[wid * round_dim + d]);
            }
        }
    }
}

void SPMM_OPT_SORT_INNERLOOP::run()
{
#ifdef BLOCK_SUMUP
    int shared_size = (WARPS_PER_BLOCK + WARPS_PER_BLOCK / 2) * DIM_MUL(dim) * sizeof(float);
#else
    int shared_size = WARPS_PER_BLOCK * DIM_MUL(dim) * sizeof(float);
#endif
    spmm_kernel_opt_sort_innerloop<<<grid, block, shared_size>>>(_block4, 0, idx, val, vin, vout, num_v, num_e, dim, 0);
}

double SPMM_OPT_SORT_INNERLOOP::do_test(bool timing)
{
    // hipMallocManaged(&coo_row, num_e * sizeof(int));
    // int k = 0;
    // for (int i = 0; i < num_v; i++)
    // {
    //     for (int j = 0; j < ptr[i + 1] - ptr[i]; j++)
    //     {
    //         coo_row[k++] = i;
    //     }
    // }

    int block_num = cuda_read_array(&this->_block4, "/home/xix22010/py_projects/graph_preprocess/block_4/" + graph + ".block4") / 4;
    if (!timing)
    {
        cout << "block num = " << block_num << endl;
    }

    grid.x = block_num;

    // block.x = DIM_MUL(dim);
    // block.y = WARPS_PER_BLOCK;
    block.x = WARPS_PER_BLOCK * 32;

    double ret = timing_body(timing);

    // hipFree(coo_row);
    hipFree(this->_block4);
    return ret;
}